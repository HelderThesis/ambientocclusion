#include "hip/hip_runtime.h"
// read_vox2gpu.cu
// Description: Import data to CPU memory, initiate 3D data array for the layered texture 
//
// TODO: check on midas + remove transform kernel

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>  // This is for automated testing output (--qatest)

//#include "read_binvox.cc"

static char *sSDKname = "AOX";

#define W 256
#define H 256
#define D 256

extern int voxels[W*H*D];

// zero copy host to device ??? good for read and/or write only once
// read from cpu to gpu 
// are we reading cpu - gpu global memory - texture memory or is it direct?

//////////////////////////////////////////////////////////////////////////////////////////////////////
// CUDA Helper Functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
  if( hipSuccess != err) {
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
        file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
  }
}

// General GPU Device CUDA Initialization
int gpuDeviceInit(int devID)
{
  int deviceCount;
  checkCudaErrors(hipGetDeviceCount(&deviceCount));
  if (deviceCount == 0) {
	fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        exit(-1);
  }
  if (devID < 0) devID = 0;
  if (devID > deviceCount-1) {
	fprintf(stderr, "\n");
        fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
        fprintf(stderr, "\n");
        return -devID;
  }

  hipDeviceProp_t deviceProp;
  checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
  if (deviceProp.major < 1) {
	fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
        exit(-1);                                                  \
  }

  checkCudaErrors( hipSetDevice(devID) );
  printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
  return devID;
}

// This function returns the best GPU (with maximum GFLOPS)
int gpuGetMaxGflopsDeviceId()
{
  int current_device   = 0, sm_per_multiproc = 0;
  int max_compute_perf = 0, max_perf_device  = 0;
  int device_count     = 0, best_SM_arch     = 0;
  hipDeviceProp_t deviceProp;

  hipGetDeviceCount( &device_count );
  // Find the best major SM Architecture GPU device
  while ( current_device < device_count ) {
	hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999) {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
   }

   // Find the best CUDA capable GPU device
   current_device = 0;
   while( current_device < device_count ) {
     hipGetDeviceProperties( &deviceProp, current_device );
     if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
	  sm_per_multiproc = 1;
     } else {
          sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
     }

     int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
     if( compute_perf  > max_compute_perf ) {
   	  // If we find GPU with SM major > 2, search only these
          if ( best_SM_arch > 2 ) {
                // If our device==dest_SM_arch, choose this, or else pass
                if (deviceProp.major == best_SM_arch) {
                     max_compute_perf  = compute_perf;
                     max_perf_device   = current_device;
                 }
           } else {
                 max_compute_perf  = compute_perf;
                 max_perf_device   = current_device;
           }
      }
       ++current_device;
    }
    return max_perf_device;
}

// Initialization code to find the best CUDA Device
int findCudaDevice(int argc, const char **argv)
{
  hipDeviceProp_t deviceProp;
  int devID = 0;
  // If the command-line has a device number specified, use it
  if (checkCmdLineFlag(argc, argv, "device")) {
  	devID = getCmdLineArgumentInt(argc, argv, "device=");
        if (devID < 0) {
           printf("Invalid command line parameters\n");
           exit(-1);
        } else {
           devID = gpuDeviceInit(devID);
           if (devID < 0) {
               printf("exiting...\n");
               shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
               exit(-1);
            }
        }
   } else {
        // Otherwise pick the device with highest Gflops/s
        devID = gpuGetMaxGflopsDeviceId();
        checkCudaErrors( hipSetDevice( devID ) );
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
   }
   return devID;
}
// end of CUDA Helper Functions

texture<float, hipTextureType2DLayered> tex; // "dim" filed in the texture reference template is now deprecated

/////////////////////////////////////////////////////////////////////////
// CUDA Kernel
// NOTE: At this point, this is just a test to ensure we are correctly saving
// Per layer: fetch layer's texture data and transform it to write to 3D output array
// NOTE: Test only! GET RID OF THIS 
__global__ void transform_kernel(float *g_odata, int width, int height, int layer) 
{
  // map from threadIdx/BlockIdx to pixel position
  unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
   
  // 0.5f offset
  float u = (x+0.5f) / (float) width;
  float v = (y+0.5f) / (float) height;

  // read from texture, do expected layered transformation and write to global memory
  g_odata[layer * width * height + y * width + x] = -tex2DLayered(tex, u, v, layer) + layer;

  //int offset = x + y * blockDim.x * gridDim.x;
  //if (cptr[offset] != 0) iptr[offset] = cptr[offset];
}

/////////////////////////////////////////////////////////////////////////
// Main
int main(int argc, char** argv) 
{
  shrQAStart(argc, argv);

  // use command-line specified CUDA device, otherwise use device with highest Gflops/s
  int devID = findCudaDevice((const int)argc, (const char **)argv);
  bool bResult = true;

  // get # SMs on GPU
  hipDeviceProp_t deviceProps;

  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s] has %d Multi-Processors ", deviceProps.name, deviceProps.multiProcessorCount );
  printf("SM %d.%d\n", deviceProps.major, deviceProps.minor );

  if (deviceProps.major < 2) {
   	printf("%s requires SM >= 2.0 to support Texture Arrays.  Test will exit... \n", sSDKname);
        hipDeviceReset();
        shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
   }

   // allocate host memory for voxel data
   unsigned int width = W, height = H, num_layers = D;
   unsigned int mem_size = width * height * num_layers * sizeof(float);
   float *voxel_data = (float*) malloc(mem_size);

   // generate input data for layered texture
   for (unsigned int layer = 0; layer < num_layers; layer++) {
   	for (int i = 0; i < (int)(width * height); i++) {
	  voxel_data[layer*width*height + i] = (float)voxels[i];
        }
   }

   // this is the expected transformation of the input data (the expected output)
   float *voxel_data_ref = (float*) malloc(size);
    for (unsigned int layer = 0; layer < num_layers; layer++)
        for (int i = 0; i < (int)(width * height); i++)
            voxel_data_ref[layer*width*height + i] = -voxel_data[layer*width*height + i] + layer;

   // allocate device memory for result
   float *d_data = NULL;
   checkCudaErrors(hipMalloc((void**) &d_data, size));   

   // allocate array and copy data
   hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
   hipArray* cu_3darray;
   checkCudaErrors( hipMalloc3DArray( &cu_3darray, &channelDesc, make_hipExtent(width, height, num_layers), hipArrayLayered ));
   hipMemcpy3DParms myparms = {0};
   myparms.srcPos = make_hipPos(0,0,0);
   myparms.dstPos = make_hipPos(0,0,0);
   myparms.srcPtr = make_hipPitchedPtr(voxel_data, width * sizeof(float), width, height);
   myparms.dstArray = cu_3darray;
   myparms.extent = make_hipExtent(width, height, num_layers);
   myparms.kind = hipMemcpyHostToDevice;
   checkCudaErrors( hipMemcpy3D( &myparms));

    // set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;  // access with normalized texture coordinates

    // bind the array to the texture
    checkCudaErrors( hipBindTextureToArray( tex, cu_3darray, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    printf("Covering 2D data array of %d x %d: Grid size is %d x %d, each block has 8 x 8 threads\n", width, height, dimGrid.x, dimGrid.y);

    transform_kernel<<< dimGrid, dimBlock >>>( d_data, width, height, 0); // warmup (for better timing)
    
    // check if kernel execution generated an error
    getLastCudaError("warmup Kernel execution failed");

    checkCudaErrors( hipDeviceSynchronize() );

    StopWatchInterface * timer;
    sdkCreateTimer( &timer );
    sdkStartTimer( &timer );

    // execute the kernel
    for (unsigned int layer = 0; layer < num_layers; layer++)
        transform_kernel<<< dimGrid, dimBlock, 0 >>>(d_data, width, height, layer);
 
    // check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors( hipDeviceSynchronize() );
    sdkStopTimer( &timer );
    printf("Processing time: %.3f msec\n", sdkGetTimerValue( &timer));
    printf("%.2f Mtexlookups/sec\n", (width*height*num_layers / (sdkGetTimerValue( &timer) / 1000.0f) / 1e6));
    sdkDeleteTimer( &timer );

    // allocate mem for the result on host side
    float* voxel_odata = (float*) malloc( size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(voxel_odata, d_data, size, hipMemcpyDeviceToHost) );

    // write regression file if necessary
    if( checkCmdLineFlag( argc, (const char **)argv, "regression") ) {
        // write file for regression test
        sdkWriteFile<float>( "./data/regression.dat", voxel_odata, width*height, 0.0f, false);
    }
    else
    {
        printf("Comparing kernel output to expected data\n");
        #define MIN_EPSILON_ERROR 5e-3f
        bResult = compareData(voxel_odata, voxel_data_ref, width * height * num_layers, MIN_EPSILON_ERROR, 0.0f);
    }

    // cleanup memory
    free(voxel_data);
    free(voxel_data_ref);
    free(voxel_odata);

    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFreeArray(cu_3darray));

    hipDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (bResult ? QA_PASSED : QA_FAILED) );
}
