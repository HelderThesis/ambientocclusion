#include "hip/hip_runtime.h"
// read_vox2gpu.cu
// Description: Import data to CPU memory, initiate 3D data array for the layered texture 
//
// TODO: check on midas + remove transform kernel - done
// TODO: distribute work accordingly blockDIm(8,8,8) screen pixel per block
// TODO: compute ao
// TODO: add timers - done
// TODO: clean up code

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <string>
#include <fstream>
#include <iostream>
#include <stdlib.h>

#include "sdkHelper.h"  // helper for shared that are common to CUDA SDK samples
#include "shrQATest.h"  // This is for automated testing output (--qatest)

#include "cuda_helpers.h"

#define TILE_SIZE 512

texture<float, hipTextureType2DLayered> tex; // "dim" filled in the texture reference template is now deprecated

/////////////////////////////////////////////////////////////////////////
// CUDA Kernel
// NOTE: At this point, this is just a test to ensure we are correctly saving
// Per layer: fetch layer's texture data and transform it to write to 3D output array
// CC: work in progress

__global__ void ao_kernel(int *devPtr, int pitch, float *c_data, int size_a) 
{
    int ty = threadIdx.y;
    int by = blockIdx.y;

    float x, y, z;
    float u, v;
    int layer;
    float dx, dy, dz;

    for (int i = 0; i <= size_a/TILE_SIZE ; i += TILE_SIZE) {
      __shared__ float bs[TILE_SIZE];
      unsigned aindex = ty + by * TILE_SIZE;

      int* row = (int*)((char*)devPtr + aindex * pitch);
      x = row[0];
      y = row[1];
      z = row[2];

      u = (x + 0.5f)/(float) 256;
      v = (y + 0.5f)/(float) 256;
      layer = (int) floor(z+0.5);
   
      if( tex2DLayered(tex, (float)u, (float)v, layer) != 0)
     	 bs[ty] = 3; 
   
      /* the output is indexed in the same manner as A therefore there is no need to duplicate x,y,z */
      //c_data[aindex] = temp; /* FOR TESTING ONLY */
    
      __syncthreads(); 
   
      // reduction of all values in bs per block 
      int i = TILE_SIZE/2;
      while (i != 0) { 
	if (ty < i) 
	   bs[ty] += bs[ty+1];
	__syncthreads();
	i /= 2;
      }
   
      // save reduced result in resultant array 
      if (ty == 0) c_data[aindex] = bs[0];
    } 
}

  // read from texture, do expected layered transformation and write to global memory
  //g_odata[layer * width * height + y * width + x] = -tex2DLayered(tex, u, v, layer) + layer;

extern "C"
void run_kernel(int width, int height, int depth, unsigned int size_a, int *a[], float *voxel_data, float *voxel_odata)
{
  // allocate device memory for A 
  int* devPtr;
  size_t pitch;
  checkCudaErrors(hipMallocPitch((void**)&devPtr, &pitch, 3 * sizeof(int), size_a));

  // copy A to allocated device memory locations
  checkCudaErrors(hipMemcpy2D(devPtr, pitch, *a, 3*sizeof(int), 3*sizeof(int), size_a, hipMemcpyHostToDevice));

  // allocate device memory for device result
  float *c = NULL;
  checkCudaErrors(hipMalloc((void**) &c, size_a * sizeof(float)));

  // allocate array and copy data
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray* cu_3darray;
  checkCudaErrors( hipMalloc3DArray( &cu_3darray, &channelDesc, make_hipExtent(width, height, depth), hipArrayLayered ));
  hipMemcpy3DParms myparms = {0};
  myparms.srcPos = make_hipPos(0,0,0);
  myparms.dstPos = make_hipPos(0,0,0);
  myparms.srcPtr = make_hipPitchedPtr(voxel_data, width * sizeof(float), width, height);
  myparms.dstArray = cu_3darray;
  myparms.extent = make_hipExtent(width, height, depth);
  myparms.kind = hipMemcpyHostToDevice;
  checkCudaErrors( hipMemcpy3D( &myparms));

  // set texture parameters
  tex.addressMode[0] = hipAddressModeWrap;
  tex.addressMode[1] = hipAddressModeWrap;
  tex.filterMode = hipFilterModeLinear;
  tex.normalized = true;  // access with normalized texture coordinates

  // bind the array to the texture
  checkCudaErrors( hipBindTextureToArray(tex, cu_3darray, channelDesc));

  StopWatchInterface * timer;
  
  // setup execution parameters
  dim3 dimBlock(TILE_SIZE,1);
  dim3 dimGrid(size_a/TILE_SIZE,1);

  checkCudaErrors(hipDeviceSynchronize());

  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  // execute the kernel
  ao_kernel <<< dimGrid, dimBlock >>> (devPtr, pitch, c, size_a); 

  // check if kernel execution generated an error
  getLastCudaError("Kernel execution failed");

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer( &timer );
  printf("Processing time: %.3f msec\n", sdkGetTimerValue( &timer));
  printf("%.2f Mtexlookups/sec\n", (width*height*depth / (sdkGetTimerValue( &timer) / 1000.0f) / 1e6));
  sdkDeleteTimer(&timer);

  // copy result from device to host
  checkCudaErrors(hipMemcpy(voxel_odata, c, size_a*sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(c));
  checkCudaErrors(hipFree(devPtr)); 
  checkCudaErrors(hipFreeArray(cu_3darray));

  return;
}

