#include "hip/hip_runtime.h"
// read_vox2gpu.cu
// Description: Import data to CPU memory, initiate 3D data array for the layered texture 
//
// TODO: check on midas + remove transform kernel - done
// TODO: distribute work accordingly blockDIm(8,8,8) screen pixel per block
// TODO: compute ao
// TODO: add timers - done
// TODO: clean up code

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <string>
#include <fstream>
#include <iostream>
#include <stdlib.h>

#include "sdkHelper.h"  // helper for shared that are common to CUDA SDK samples
#include "shrQATest.h"  // This is for automated testing output (--qatest)

#include "cuda_helpers.h"

#define TILE_SIZE 512//64

texture<float, hipTextureType2DLayered> tex; // "dim" filled in the texture reference template is now deprecated

/////////////////////////////////////////////////////////////////////////
// CUDA Kernel
// NOTE: At this point, this is just a test to ensure we are correctly saving
// Per layer: fetch layer's texture data and transform it to write to 3D output array
// CC: work in progress

__global__ void test0_kernel(int *devPtr, int pitch, float *c_data, int size_a) 
{
  int ty = threadIdx.x;
  int by = blockIdx.x;

  float x, y, z;

  unsigned aindex = ty + by * TILE_SIZE;

  int* row = (int*)((char*)devPtr + aindex * pitch);
  x = row[0];
  y = row[1];
  z = row[2];
  
  c_data[aindex] = x + y + z; 
}

__global__ void test1_kernel(int *ad1, int* ad2, int *ad3, float *c_data, int size_a) 
{
  int ty = threadIdx.x;
  int by = blockIdx.x;
  unsigned aindex = ty + by * TILE_SIZE;

  c_data[aindex] = ad1[aindex] + ad2[aindex] + ad3[aindex];
}

__global__ void test2_kernel(int *ad1, int* ad2, int *ad3, float *c_data, int size_a) 
{
  int ty = threadIdx.x;
  int by = blockIdx.x;
  float u, v;
  int layer;
  unsigned aindex = ty + by * TILE_SIZE;

  u = (ad1[aindex] + 0.5f)/(float) 256;
  v = (ad2[aindex] + 0.5f)/(float) 256;
  layer = (int) floor(ad3[aindex] + 0.5);
   
  c_data[aindex] = tex2DLayered(tex, (float)u, (float)v, layer);
}

__global__ void test3_kernel(int *ad1, int *ad2, int *ad3, float *c_data, int size_a)
{
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  float u, v;
  float v_top, v_bottom, u_right, u_left;
  int layer;
  float temp;
  unsigned aindex = tx + bx * TILE_SIZE;

  u = (ad1[aindex] + 0.5f)/(float) 256;
  v = (ad2[aindex] + 0.5f)/(float) 256;
  layer = (int) floor(ad3[aindex] + 0.5);

  v_top = (ad2[aindex] - 1.0f + 0.5f)/(float) 256;
  v_bottom = (ad2[aindex] + 1.0f + 0.5f)/(float) 256;
  u_right = (ad1[aindex] - 1.0f + 0.5f)/(float) 256;
  u_left = (ad1[aindex] + 1.0f + 0.5f)/(float) 256;

  if (v_top>=0 && v_bottom<=1 && u_right>=0 && u_left<=1) { 
  //  c_data[aindex] = tex2DLayered(tex, (float)u, (float)v, layer) + tex2DLayered(tex, (float)u, (float)v_top, layer) + tex2DLayered(tex, (float)u, (float)v_bottom, layer) + tex2DLayered(tex, (float)u_left, (float)v, layer) + tex2DLayered(tex, (float)u_right, (float)v, layer);
  
  c_data[aindex] = tex2DLayered(tex, (float)u, (float)v_top, layer) * powf(ad2[aindex] - 1.0f - ad2[aindex],2) + tex2DLayered(tex, (float)u, (float)v_bottom, layer) * powf(ad2[aindex] + 1.0f - ad2[aindex],2) + tex2DLayered(tex, (float)u_left, (float)v, layer) * powf(ad1[aindex] + 1.0f - ad1[aindex],2) + tex2DLayered(tex, (float)u_right, (float)v, layer) * powf(ad1[aindex] - 1.0f - ad1[aindex],2);
  }
  else c_data[aindex] = 0;
}

__global__ void test4_kernel(int *ad1, int *ad2, int *ad3, float *c_data, int size_a)
{
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  float u, v;
  int layer;
  float u_next, v_next;
  int layer_next;
  float temp = 0;
  float dx = 0.0f; float dy = 0.0f; float dz = 0.0f;
  unsigned aindex = tx + bx * TILE_SIZE;
  int dim = 4;
  int exist; 

  u = (ad1[aindex] )/(float) 255;
  v = (ad2[aindex] )/(float) 255;
  layer = ad3[aindex];

  float mine = tex2DLayered(tex, (float)u, (float)v, layer);

  if (mine != 0) {
  for (int m=-(dim/2); m<=(dim/2)-1; m++) {                    // depth
      for (int n=-(dim/2)+1; n<=(dim/2); n++) {                // height
        for (int p=-(dim/2); p<=(dim/2)-1; p++) {              // width
	   u_next = (ad1[aindex] + p)/(float) 255;
	   v_next = (ad2[aindex] + n)/(float) 255;
	   layer_next = ad3[aindex] + m;
	   exist = tex2DLayered(tex, (float)u_next, (float)v_next, layer_next);
           if ( exist!=0 && v_next>=0 && v_next<=1 && u_next>=0 && u_next<=1 && layer_next>=0 && layer_next<256) { 
           	dx = powf(p,2);
           	dy = powf(n,2);
           	dz = powf(m,2);
           	temp += dx + dy + dz;
	   } 
	   else temp += 0;
        }
     }
  }
  }
  
  c_data[aindex] = temp;
  __syncthreads();
}

__global__ void ao_kernel(int *ad1, int* ad2, int*ad3, float *c_data, int size_a) 
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;

    float x, y, z;
    float u, v;
    int layer;
    float dx, dy, dz;
    __shared__ float bs[TILE_SIZE];
    unsigned int aindex = tx + bx * TILE_SIZE;  

    c_data[tx + bx * TILE_SIZE] = 0;
  

    //for (int i = 0; i <= size_a/TILE_SIZE ; i += TILE_SIZE) {
    //  __shared__ float bs[TILE_SIZE];
    //  unsigned aindex = tx + bx * TILE_SIZE;

      x = ad1[aindex];
      y = ad2[aindex];
      z = ad3[aindex];

      u = (x + 0.5f)/(float) 256;
      v = (y + 0.5f)/(float) 256;
      layer = (int) floor(z+0.5);
   
      if( tex2DLayered(tex, (float)u, (float)v, layer) != 0)
     	 bs[tx] = 3;
 
      /* the output is indexed in the same manner as A therefore there is no need to duplicate x,y,z */
      //c_data[aindex] = temp; /* FOR TESTING ONLY */
    
      __syncthreads(); 
   
      // reduction of all values in bs per block 
      int i = TILE_SIZE/2;
      while (i != 0) { 
	if (tx < i) 
	   bs[tx] += bs[tx+1];
	__syncthreads();
	i /= 2;
      }
   
      // save reduced result in resultant array 
      if (tx == 0) c_data[bx] = bs[0]; 
}

  // read from texture, do expected layered transformation and write to global memory
  //g_odata[layer * width * height + y * width + x] = -tex2DLayered(tex, u, v, layer) + layer;

extern "C"
void run_kernel(int width, int height, int depth, unsigned int size_a, int *a[], float *voxel_data, float *voxel_odata)
{
  // allocate device memory for A 
  int* devPtr;
  size_t pitch;
  checkCudaErrors(hipMallocPitch((void**)&devPtr, &pitch, 3 * sizeof(int), size_a));
  // copy A to allocated device memory locations
  checkCudaErrors(hipMemcpy2D(devPtr, pitch, (void *)a, 3*sizeof(int), 3*sizeof(int), size_a, hipMemcpyHostToDevice));

/*TEMPORARY HACK because the above does not work as expected
  We split "a" in 3 1d arrays d1,d2,d3 and we copyone-by-one in device memory
 */
  int *d1 = NULL;
  int *ad1 = NULL;
  d1 = (int*) malloc(size_a * sizeof(int));
  for (int i=0; i<size_a; i++)
    d1[i]=a[i][0];
  checkCudaErrors(hipMalloc((void**) &ad1, size_a * sizeof(int)));
  checkCudaErrors(hipMemcpy(ad1, d1, size_a * sizeof(int), hipMemcpyHostToDevice));

  int *d2 = NULL;
  int *ad2 = NULL;
  d2 = (int*) malloc(size_a * sizeof(int));
  for (int i=0; i<size_a; i++)
    d2[i]=a[i][1];
  checkCudaErrors(hipMalloc((void**) &ad2, size_a * sizeof(int)));
  checkCudaErrors(hipMemcpy(ad2, d2, size_a * sizeof(int), hipMemcpyHostToDevice));

  int *d3 = NULL;
  int *ad3 = NULL;
  d3 = (int*) malloc(size_a * sizeof(int));
  for (int i=0; i<size_a; i++)
    d3[i]=a[i][2];
  checkCudaErrors(hipMalloc((void**) &ad3, size_a * sizeof(int)));
  checkCudaErrors(hipMemcpy(ad3, d3, size_a * sizeof(int), hipMemcpyHostToDevice));
/* END of HACK */

  // allocate device memory for device result
  float *c = NULL;
  checkCudaErrors(hipMalloc((void**) &c, size_a * sizeof(float)));

  // allocate array and copy data
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray* cu_3darray;
  checkCudaErrors( hipMalloc3DArray( &cu_3darray, &channelDesc, make_hipExtent(width, height, depth), hipArrayLayered ));
  hipMemcpy3DParms myparms = {0};
  myparms.srcPos = make_hipPos(0,0,0);
  myparms.dstPos = make_hipPos(0,0,0);
  myparms.srcPtr = make_hipPitchedPtr(voxel_data, width * sizeof(float), width, height);
  myparms.dstArray = cu_3darray;
  myparms.extent = make_hipExtent(width, height, depth);
  myparms.kind = hipMemcpyHostToDevice;
  checkCudaErrors( hipMemcpy3D( &myparms));

  // set texture parameters
  tex.addressMode[0] = hipAddressModeWrap;
  tex.addressMode[1] = hipAddressModeWrap;
  tex.filterMode = hipFilterModeLinear;
  tex.normalized = true;  // access with normalized texture coordinates

  // bind the array to the texture
  checkCudaErrors( hipBindTextureToArray(tex, cu_3darray, channelDesc));

  StopWatchInterface * timer;
  
  // setup execution parameters
  dim3 dimBlock(TILE_SIZE);
  dim3 dimGrid((size_a)/TILE_SIZE+1);

  checkCudaErrors(hipDeviceSynchronize());

  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  // execute the kernel
//  ao_kernel <<< dimGrid, dimBlock >>> (ad1, ad2, ad3, c, size_a); 
//  test0_kernel <<< dimGrid, dimBlock >>> (devPtr, pitch, c, size_a); 
//  test1_kernel <<< dimGrid, dimBlock >>> (ad1, ad2, ad3, c, size_a); 
//  test2_kernel <<< dimGrid, dimBlock >>> (ad1, ad2, ad3, c, size_a); 
//  test3_kernel <<< dimGrid, dimBlock >>> (ad1, ad2, ad3, c, size_a);
  test4_kernel <<< dimGrid, dimBlock >>> (ad1, ad2, ad3, c, size_a);
 
  // check if kernel execution generated an error
  getLastCudaError("Kernel execution failed");

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer( &timer );
  printf("Processing time: %.3f msec\n", sdkGetTimerValue( &timer));
  printf("%.2f Mtexlookups/sec\n", (width*height*depth / (sdkGetTimerValue( &timer) / 1000.0f) / 1e6));
  sdkDeleteTimer(&timer);

  // copy result from device to host
  checkCudaErrors(hipMemcpy(voxel_odata, c, size_a*sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(ad1));
  checkCudaErrors(hipFree(ad2));
  checkCudaErrors(hipFree(ad3));
  checkCudaErrors(hipFree(c));
  checkCudaErrors(hipFree(devPtr)); 
  checkCudaErrors(hipFreeArray(cu_3darray));

  return;
}

